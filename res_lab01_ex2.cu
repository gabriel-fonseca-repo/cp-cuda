
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define BLOCK_SIZE 64
#define ARRAY_SIZE 128000

typedef struct timeval tval;

float generate_hash(int n, float *y) {
    float hash = 0.0f;

    for (int i = 0; i < n; i++) {
        hash += y[i];
    }

    return hash;
}

double get_elapsed(tval t0, tval t1) {
    return (double)(t1.tv_sec - t0.tv_sec) * 1000.0L +
           (double)(t1.tv_usec - t0.tv_usec) / 1000.0L;
}

void cpu_saxpy(int n, float a, float *x, float *y) {
    for (int i = 0; i < n; i++) {
        y[i] = a * x[i] + y[i];
    }
}

// TO-DO #2.1
// Declarando e implementando o kernel CUDA executará na GPU.
__global__ void gpu_saxpy(int n, float a, float *x, float *y) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < n) {
        y[tid] = a * x[tid] + y[tid];
    }
}

int main(int argc, char **argv) {
    float a = 0.0f;
    float *x = NULL;
    float *y = NULL;
    float error = 0.0f;

    // Verifique se a contante foi fornecida
    // if (argc != 2) {
    //     fprintf(stderr, "Erro: A constante está faltando!\n");
    //     return -1;
    // }

    // Código original do problema SAXPY
    a = 10;
    x = (float *)malloc(sizeof(float) * ARRAY_SIZE);
    y = (float *)malloc(sizeof(float) * ARRAY_SIZE);
    for (int i = 0; i < ARRAY_SIZE; i++) {
        x[i] = 0.1f;
        y[i] = 0.2f;
    }

    // TO-DO #2.2
    // Definindo a distribuição das threads, em termos da dimensão da grade
    // (grid) e da dimensão de cada bloco (de threads) dentro da grade.
    dim3 grid((ARRAY_SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE);
    dim3 block(BLOCK_SIZE);

    // TO-DO #2.3.1
    // Declarando e definindo a memória GPU necessária para executar o kernel
    // CUDA.
    float *d_x, *d_y;
    hipMalloc((void **)&d_x, sizeof(float) * ARRAY_SIZE);
    hipMalloc((void **)&d_y, sizeof(float) * ARRAY_SIZE);

    // TO-DO #2.3.2
    // Transferindo os dados do host para a GPU.
    hipMemcpy(d_x, x, sizeof(float) * ARRAY_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, sizeof(float) * ARRAY_SIZE, hipMemcpyHostToDevice);

    // Variáveis para métricas de performance/tempo de execução da CPU.
    tval t_start, t_end;
    gettimeofday(&t_start, NULL);
    // Execução na CPU para fins de comparação
    cpu_saxpy(ARRAY_SIZE, a, x, y);
    gettimeofday(&t_end, NULL);
    double elapsed_cpu = get_elapsed(t_start, t_end);
    error = generate_hash(ARRAY_SIZE, y);

    printf("Execução na CPU terminada. Taxa de erro = %.6f.\n", error);
    printf("Tempo de execução na CPU: %.6f ms.\n", elapsed_cpu);

    // Variáveis para métricas de performance/tempo de execução da GPU.
    tval t_start_gpu, t_end_gpu;
    gettimeofday(&t_start_gpu, NULL);

    // TO-DO #2.4
    // Execute o kernel CUDA com os parâmetros correspondentes.
    gpu_saxpy<<<grid, block>>>(ARRAY_SIZE, a, d_x, d_y);
    hipDeviceSynchronize();

    gettimeofday(&t_end_gpu, NULL);
    double elapsed_gpu = get_elapsed(t_start_gpu, t_end_gpu);

    // TO-DO #2.5.1
    // Transferindo os resultados da GPU para o host.
    hipMemcpy(y, d_y, sizeof(float) * ARRAY_SIZE, hipMemcpyDeviceToHost);

    error = fabsf(error - generate_hash(ARRAY_SIZE, y));

    printf("Execução na GPU terminada. Taxa de erro = %.6f.\n", error);
    printf("Tempo de execução na GPU: %.6f ms.\n", elapsed_gpu);

    if (error > 0.0001f) {
        fprintf(stderr, "Erro: a solução está incorreta!\n");
    }

    // Gerenciamento de memória.
    free(x);
    free(y);
    hipFree(d_x);
    hipFree(d_y);

    return 0;
}